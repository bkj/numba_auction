#undef _GLIBCXX_ATOMIC_BUILTINS
#undef _GLIBCXX_USE_INT128 

#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <iostream>

#include <chrono>
#include <random>

using namespace std::chrono;

#define SEED 123123
std::default_random_engine generator(SEED);

const int n_bidders = 20000;
const int n_items   = 20000;
const int max_cost  = 10;

void uniform_random_problem(int* cost_matrix) {
  std::uniform_int_distribution<int> distribution(0, max_cost);

  for (long i = 0; i < n_items * n_bidders; i++) {
    cost_matrix[i] = distribution(generator);
  }
}

struct GpuTimer {
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer() {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer() {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start() { hipEventRecord(start, 0); }
    void Stop()  { hipEventRecord(stop, 0);  }

    float ElapsedMillis() {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

// C-style indexing
int ci(int row, int column, int nColumns) {
  return row*nColumns+column;
}

// Convert a linear index to a row index
template <typename T>
struct linear_index_to_row_index : public thrust::unary_function<T,T>
{
  T C; // number of columns
  
  __host__ __device__
  linear_index_to_row_index(T C) : C(C) {}

  __host__ __device__
  T operator()(T i) {
    return i / C;
  }
};

typedef thrust::tuple<int,int> OpType;

struct Op : public thrust::binary_function<OpType,OpType,OpType> {
    __host__ __device__
        OpType operator()(const OpType& a, const OpType& b) const {
          if (thrust::get<1>(a) > thrust::get<1>(b)){
            return a;
          } else {
            return b;
          }
        }
};

thrust::device_vector<OpType> compute_mins(thrust::device_vector<int> A, int nRows, int nColumns) {
  // allocate storage for row Ops and indices
  thrust::device_vector<OpType> results(nRows);
  thrust::device_vector<int> indices(nRows);          
      
  // compute row Ops by finding Op values with equal row indices
  thrust::reduce_by_key
    (thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(nColumns)),
     thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(nColumns)) + (nRows*nColumns),
     thrust::make_zip_iterator(thrust::make_tuple(thrust::counting_iterator<int>(0),A.begin())),
     indices.begin(),
     results.begin(),
     thrust::equal_to<int>(),
     Op());
    
  return results;
}

int main(int argc, char *argv[]) {
  int* A = (int*)malloc(n_items * n_bidders * sizeof(int));
  uniform_random_problem(A);
  std::cout << "uniform_random_problem: done" << std::endl;
  
  // --
  // CPU
  
  auto t_start = high_resolution_clock::now();
  
  int* acc = (int*)malloc(n_bidders * sizeof(int));
  for(int i = 0; i < n_bidders; i++) {
    int min_val = -1;
    for(int j = 0; j < n_items; j++) {
      int val = A[i * n_items + j];
      if(val > min_val) {
        min_val = val;
      }
    }    
    acc[i] = min_val;
  }
  
  auto t_stop = high_resolution_clock::now();
  auto cpu_elapsed = duration_cast<microseconds>(t_stop - t_start).count();
  
  // Inspect
  int cpu_acc = 0;
  for(int i = 0 ; i < n_bidders ; i++) {
    cpu_acc += acc[i];
  }
  
  // --
  // GPU
    
  thrust::host_vector<int> B(A, A + (n_items * n_bidders));
  thrust::device_vector<int> C = B;
  
  // Warmup
  thrust::device_vector<OpType> dummy_results = compute_mins(C, n_bidders, n_items);
  
  GpuTimer timer = GpuTimer();
  timer.Start();
  
  thrust::device_vector<OpType> results = compute_mins(C, n_bidders, n_items);
  
  timer.Stop();
  float gpu_elapsed = timer.ElapsedMillis();

  // Inspect
  thrust::host_vector<OpType> h_results = results;
  int gpu_acc = 0;
  for(int i = 0; i < n_bidders; i++) {
    // std::cout 
    //   << thrust::get<0>(h_results[i]) / 10 << " " 
    //   << thrust::get<0>(h_results[i]) % 10 << " " 
    //   << thrust::get<1>(h_results[i]) << std::endl;
    gpu_acc += thrust::get<1>(h_results[i]);
  }
    
  printf("gpu_elapsed=%f | cpu_elapsed=%f | gpu_acc=%d | cpu_acc=%d \n", gpu_elapsed, (float)cpu_elapsed / 1000, gpu_acc, cpu_acc);
  
  return 0;
  
  // GPU is ~ 10x faster for sufficiently large matrices.  However, doesn't seem to make a ton of different until > 5K or so.
  // Still ... could be useful
}
